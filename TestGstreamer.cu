#include "hip/hip_runtime.h"
﻿//for library use
#include <fstream>
#include <ostream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

//link with other reader and writer
#include "GstreamerReader.h"
#include "GstreamerWriter.h"


//some settings:
const int COLOR_CHANNELS=3;
const int BLOCK_SIZE=32;
//define a 3x3 matrix now is a 1d array, so the length is 3*3, and to avoid passing length from gpu, use a global
const int TRANSFORMATION_MATRIX_SIZE = 3 * 3;
//the 3x3 transmation matrix now is saved to be 1d array for better efficient in CUDA
float h_transformationMatrix_BT2020toBT709[TRANSFORMATION_MATRIX_SIZE] = {
    1.660491f, -0.587641f, -0.072850f,
    -0.124550f, 1.132900f, -0.008349f,
    -0.018151f, -0.100579f, 1.118730f
};



// CUDA kernel for color space conversion
__global__ void color_transformation_3x3(uchar3* src, uchar3* dst, int width, int height, const float* transformationMatrix) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int index = y * width + x;

    float R2020 = src[index].x;
    float G2020 = src[index].y;
    float B2020 = src[index].z;

    // Example conversion formulas
    float R709 = R2020 * transformationMatrix[0] + G2020 * transformationMatrix[1] + B2020 * transformationMatrix[2];
    float G709 = R2020 * transformationMatrix[3] + G2020 * transformationMatrix[4] + B2020 * transformationMatrix[5];
    float B709 = R2020 * transformationMatrix[6] + G2020 * transformationMatrix[7] + B2020 * transformationMatrix[8];

    dst[index].x = fminf(fmaxf(R709, 0.0f), 255.0f);
    dst[index].y = fminf(fmaxf(G709, 0.0f), 255.0f);
    dst[index].z = fminf(fmaxf(B709, 0.0f), 255.0f);
}


// a class do frame color conversion
class CudaFrameConverter {

private:
    uchar3* d_src = nullptr;
    uchar3* d_dst = nullptr;
    float* d_transformationMatrix = nullptr; 
    int frameWidth, frameHeight;
    size_t frameSize;


public:
    //assigning cuda memory when this object is called, then there are no copy memory used
    CudaFrameConverter(int width, int height, const float* h_transformationMatrix)  : frameWidth(width), frameHeight(height) {
        
        frameSize = width * height * COLOR_CHANNELS;

        //this is the setting for the convert objects, make sure they are load once
        hipMalloc(&d_src, frameSize);
        hipMalloc(&d_dst, frameSize);

        //also load for the transformation matrix here
        hipMalloc(&d_transformationMatrix, TRANSFORMATION_MATRIX_SIZE * sizeof(float)); // Allocate memory for the 3x3 matrix
        hipMemcpy(d_transformationMatrix, h_transformationMatrix, TRANSFORMATION_MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice); // Copy matrix to device
    
    }

    //for destructor clean the memory
    ~CudaFrameConverter() {
        hipFree(d_src);
        hipFree(d_dst);
        hipFree(d_transformationMatrix); 

    }



    void ConvertFrame(const std::vector<unsigned char>& inputFrame, std::vector<unsigned char>& outputFrame) {
        
        hipMemcpy(d_src, inputFrame.data(), frameSize, hipMemcpyHostToDevice);
        // Perform color space conversion

        // Adjust gridSize and blockSize as necessary
        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE); 
        dim3 gridSize((frameWidth + blockSize.x - 1) / blockSize.x, (frameHeight + blockSize.y - 1) / blockSize.y);
        color_transformation_3x3<<<gridSize, blockSize>>>(d_src, d_dst, frameWidth, frameHeight, d_transformationMatrix);
        hipMemcpy(outputFrame.data(), d_dst, frameSize, hipMemcpyDeviceToHost);
    
    }

};




void TestVideo(const std::string& inputUrl, const std::string& outUrl  ) {
    
    
    // load the reader
    GstreamerReader reader;

    // check if address is valid
    if (reader.Open(inputUrl) < 0) {
        std::cerr << "Failed to open video: " << inputUrl << std::endl;
        return;
    }

    // get the width and height of the video after opened the video
    int videoWidth = reader.GetWidth();
    int videoHeight = reader.GetHeight();


    // check if the height and width are valid
    if (videoWidth <= 0 || videoHeight <= 0) {
        std::cerr << "Invalid video dimensions." << std::endl;
        return;
    }

    //set reader size
    reader.InputOriginSize(videoWidth, videoHeight);
    // end of loading reader




    // load the writer
    GstreamerWriter writer;

    //use reader info to set reader
    writer.SetSize(videoWidth, videoHeight);
    writer.SetFramerate(reader.Framerate());

    // check of writer is valid
    if (writer.Open(outUrl) < 0){
        return;}

    //end of writer


    //call the convert object to convert color for each frame
    CudaFrameConverter converter(videoWidth, videoHeight, h_transformationMatrix_BT2020toBT709);

    //buffer size calculated
    std::vector<unsigned char> frameBuffer, convertedFrame(videoWidth* videoHeight * COLOR_CHANNELS); 
    
    //initialize the tempstamp
    double timestamp = .0;

    //to do image processing, call the memory for parameter outside of loop

    //when there is frame is the buffer, then keep running
    while (reader.Read(frameBuffer, timestamp) >= 0) {
        
        converter.ConvertFrame(frameBuffer, convertedFrame);

        writer.Write(convertedFrame, timestamp);
    }

    //std::cout << "video read and write finished." << std::endl;

}




int main(int argc, char* argv[]) {

    //check if number of parameter correct:
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <inputUrl> <outputUrl>" << std::endl;
        return -1;
    }

    //if it is correct
	gst_init(&argc, &argv);

	std::string inputUrl(argv[1]);
	std::string outputUrl(argv[2]);
	// std::cout << "read video:" << inputUrl << std::endl;
	
    //handling address check in TestVideo function
    TestVideo(inputUrl, outputUrl);
	
    return 0;
}

